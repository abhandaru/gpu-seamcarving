#include "hip/hip_runtime.h"
//
// 18645 - GPU Seamcarving
// Authors: Adu Bhandaru, Matt Sarett
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "energies.h"

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32

using std::cout;
using std::endl;
using std::min;


//
// Kernel functions.
//

__global__
void compute_energies_kernel(RGBQuad* image, float* energies,
    int width, int height) {
  // Extract thread and block index information
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;
  int row = by * BLOCK_HEIGHT + ty;
  int col = bx * BLOCK_WIDTH + tx;

  // Allocate shared memory with padding to avoid bank conflicts
  __shared__ RGBQuad shared_image[BLOCK_WIDTH * (BLOCK_WIDTH + 1)];

  // Populate shared memory with image pixels
  if (row < height && col < width) {
    shared_image[ty * (BLOCK_WIDTH + 1) + tx] = image[row * width + col];
  }
  else {
    RGBQuad& pixel = shared_image[ty * (BLOCK_WIDTH + 1) + tx];
    pixel.red = 0;
    pixel.green = 0;
    pixel.blue = 0;
    //return;
  }

  // Wait for all threads to finish loading shared memory
  __syncthreads();

  // Get current pixel
  RGBQuad& current = shared_image[ty * (BLOCK_WIDTH + 1) + tx];

  // Declare difference
  float dx2;
  float dy2;

  // Compute differences with boundary checks
  if (col == width - 1) {
    float dx_red = (float)current.red;
    float dx_green = (float)current.green;
    float dx_blue = (float)current.blue;
    dx2 = dx_red * dx_red + dx_green * dx_green + dx_blue * dx_blue;
  }
  else if (tx == BLOCK_WIDTH - 1) {
    RGBQuad& right = image[row * width + col + 1];
    float dx_red = (float)right.red - current.red;
    float dx_green = (float)right.green - current.green;
    float dx_blue = (float)right.blue - current.blue;
    dx2 = dx_red * dx_red + dx_green * dx_green + dx_blue * dx_blue;
  }
  else {
    RGBQuad& right = shared_image[ty * (BLOCK_WIDTH + 1) + tx + 1];
    float dx_red = (float)right.red - current.red;
    float dx_green = (float)right.green - current.green;
    float dx_blue = (float)right.blue - current.blue;
    dx2 = dx_red * dx_red + dx_green * dx_green + dx_blue * dx_blue;
  }

  if (row == height - 1) {
    float dy_red = (float)current.red;
    float dy_green = (float)current.green;
    float dy_blue = (float)current.blue;
    dy2 = dy_red * dy_red + dy_green * dy_green + dy_blue * dy_blue;
  }
  else if (ty == BLOCK_HEIGHT - 1) {
    RGBQuad& down = image[(row + 1) * width + col];
    float dy_red = (float)down.red - current.red;
    float dy_green = (float)down.green - current.green;
    float dy_blue = (float)down.blue - current.blue;
    dy2 = dy_red * dy_red + dy_green * dy_green + dy_blue * dy_blue;
  }
  else {
    RGBQuad& down = shared_image[(ty + 1) * (BLOCK_WIDTH + 1) + tx];
    float dy_red = (float)down.red - current.red;
    float dy_green = (float)down.green - current.green;
    float dy_blue = (float)down.blue - current.blue;
    dy2 = dy_red * dy_red + dy_green * dy_green + dy_blue * dy_blue;
  }

  // Compute gradient
  float grad = (float) sqrt(dx2 + dy2);

  // Store results
  int index = row * width + col;
  energies[index] = grad;
}


//
// Class methods.
//

Energies::Energies(Image* image) {
  _width = image->width();
  _height = image->height();
  _image = image;
  _energies = new float[_width * _height];
}


Energies::~Energies() {
  delete _energies;
}


//
// Filtering options
// We may need to compute the grayscale values first.
// See this article: http://stackoverflow.com/a/15686412/408940
//
void Energies::compute() {

  // Declare pointers for device memory
  RGBQuad* image_d;
  float* energies_d;
  int image_size = _width * _height * sizeof(RGBQuad);
  int energies_size = _width * _height * sizeof(float);

  // Allocate device memory and for inputs and outputs
  hipMalloc((void**) &image_d, image_size);
  hipMemcpy(image_d, _image->getPixels(), image_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &energies_d, energies_size);

  // Invoke the kernel to compute the energies
  int num_blocks_x = (_width - 1) / BLOCK_WIDTH + 1;
  int num_blocks_y = (_height - 1) / BLOCK_HEIGHT + 1;
  int num_threads_x = min(BLOCK_WIDTH, _width);
  int num_threads_y = min(BLOCK_HEIGHT, _height);
  dim3 dim_grid(num_blocks_x, num_blocks_y, 1);
  dim3 dim_block(num_threads_x, num_threads_y, 1);
  compute_energies_kernel<<<dim_grid, dim_block>>>
      (image_d, energies_d, _width, _height);

  // Transfer result from device to host
  hipMemcpy(_energies, energies_d, energies_size, hipMemcpyDeviceToHost);
  hipFree(image_d);
  hipFree(energies_d);
}


//
// Getters and operators
//
int Energies::width() const {
  return _width;
}


int Energies::height() const {
  return _height;
}


float Energies::get(int row, int col) const {
  if (row < 0 || row >= _height ||
      col < 0 || col >= _width) {
    // Return a high value so this is never the min.
    return MAX_VALUE;
  }

  // Common case.
  int index = row * _width + col;
  return _energies[index];
}


float* Energies::getEnergies() const {
  return _energies;
}


void Energies::set(int row, int col, float value) {
  int index = row * _width + col;
  _energies[index] = value;
}


void Energies::print() const {
  cout << "Energies: " << endl;
  for (int i = 0; i < _height; i++) {
    for (int j = 0; j < _width; j++) {
      cout << get(i, j) << "\t";
    }
    cout << endl;
  }
  cout << endl;
}


const float* Energies::operator [](int i) const {
  return _energies + (i * _width);
};
