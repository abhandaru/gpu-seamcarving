#include "hip/hip_runtime.h"
//
// 18645 - GPU Seamcarving
// Authors: Adu Bhandaru, Matt Sarett
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "seamcarver.h"

#define MAX_THREADS 1024

using std::cout;
using std::endl;
using std::min;
using std::vector;


//
// Kernel functions.
//

__global__
static void find_min_kernel(float* row, float* mins, int* min_indices,
    int width, int power) {
  // Compute current index.
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int index = tx + bx * MAX_THREADS;

  // Set up shared memory for tracking mins.
  extern __shared__ float shared_memory[];
  float* shared_mins = (float*) shared_memory;
  int* shared_min_indices = (int*) (&(shared_memory[power]));

  // Copy global intermediate values into shared memory.
  shared_mins[tx] = (index < width) ? row[index] : MAX_VALUE;
  shared_min_indices[tx] = (index < width) ? index : -1;
  __syncthreads();

  // Do the reduction for value pairs.
  for (int i = power / 2; i > 0; i >>= 1) {
    if (tx < i) {
      if (shared_mins[tx] > shared_mins[tx + i]) {
        shared_mins[tx] = shared_mins[tx + i];
        shared_min_indices[tx] = shared_min_indices[tx + i];
      }
    }
    __syncthreads();
  }

  // Thread 0 has the solution.
  if (tx == 0) {
    mins[bx] = shared_mins[0];
    min_indices[bx] = shared_min_indices[0];
  }
}


__global__
void compute_min_cost_kernel(float* energies, float* min_costs,
    int width, int height) {
  // Extract thread and block index information
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int col = bx * MAX_THREADS + tx;

  // Allocate shared memory with padding to avoid bank conflicts
  __shared__ float shared_costs[MAX_THREADS];

  // Load the first row of shared memory with energies and min costs
  if (col < width) {
    shared_costs[tx] = energies[col];
    min_costs[col] = energies[col];
  } else {
    return;
  }

  // Wait for all threads to finish loading the first row of shared memory.
  __syncthreads();

  // Compute minimum costs row by row w/ by double buffering.
  for (int row = 1; row < height; row++) {
    float left = (tx > 0) ? shared_costs[tx - 1] : MAX_VALUE;
    float middle = shared_costs[tx];
    float right = (tx < width - 1) ? shared_costs[tx + 1] : MAX_VALUE;

    // Compute the minimum and then add cost of current cell
    float minimum = min(left, min(middle, right));
    float cost = minimum + energies[row * width + col];
    __syncthreads();
    shared_costs[tx] = cost;
    __syncthreads();
    min_costs[row * width + col] = cost;
  }
}


//
// Class methods.
//

Seamcarver::Seamcarver(Image* image) {
  _image = image;
}


Seamcarver::~Seamcarver() {

}


// Simply remove n seams.
void Seamcarver::removeSeams(int n) {
  for (int i = 0; i < n; i++) {
    removeSeam();
  }
}


// Removes 1 seam.
void Seamcarver::removeSeam() {
  findSeam();
  _image->removeSeam(_seam);
}


// Finds the seam of the lowest cost.
void Seamcarver::findSeam() {
  Energies energies(_image);
  energies.compute();
  float* energies_h = energies.getEnergies();

  // Declare pointers for device and host memory
  float* energies_d;
  float* min_cost_d;
  int width = energies.width();
  int height = energies.height();
  int size = width * height * sizeof(float);

  // Allocate device memory and for inputs and outputs
  hipMalloc((void**) &energies_d, size);
  hipMemcpy(energies_d, energies_h, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &min_cost_d, size);

  // Invoke the kernel to compute the min cost table
  int num_blocks = (width - 1) / MAX_THREADS + 1;
  int num_threads = min(MAX_THREADS, width);
  dim3 dim_grid(num_blocks, 1, 1);
  dim3 dim_block(num_threads, 1, 1);
  compute_min_cost_kernel<<<dim_grid, dim_block>>>
      (energies_d, min_cost_d, width, height);

  // Transfer result from device to host
  hipMemcpy(energies_h, min_cost_d, size, hipMemcpyDeviceToHost);
  hipFree(energies_d);
  hipFree(min_cost_d);

  // Calculate threads and blocks for a minimum reduction
  num_threads = min(nextPower2(width), MAX_THREADS);
  num_blocks = (width - 1) / num_threads + 1;
  int row_size = width * sizeof(float);
  int mins_size = num_blocks * sizeof(float);
  int min_indices_size = num_blocks * sizeof(int);
  int shared_size = num_threads * (sizeof(float) + sizeof(int));

  // Declare pointers for device and host memory
  float* row = &(energies_h[(height - 1) * width]);
  float* mins = (float*) malloc(mins_size);
  int* min_indices = (int*) malloc(min_indices_size);
  float* row_d;
  float* mins_d;
  int* min_indices_d;
  hipMalloc((void**) &row_d, row_size);
  hipMemcpy(row_d, row, row_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &mins_d, mins_size);
  hipMalloc((void**) &min_indices_d, mins_size);

  // Use the kernel function to find intermediate minimums
  find_min_kernel<<<num_blocks, num_threads, shared_size>>>
      (row_d, mins_d, min_indices_d, width, num_threads);

  // Compute final minimum
  hipMemcpy(mins, mins_d, mins_size, hipMemcpyDeviceToHost);
  hipMemcpy(min_indices, min_indices_d, min_indices_size,
    hipMemcpyDeviceToHost);
  float minimum = mins[0];
  int min_index = min_indices[0];
  for (int i = 1; i < num_blocks; i++) {
    if (mins[i] < minimum) {
      minimum = mins[i];
      min_index = min_indices[i];
    }
  }

  // Create the seam in reverse order.
  _seam.clear();
  _seam.push_back(min_index);
  for (int i = height - 2; i >= 0; i--) {
    float left = energies.get(i, min_index - 1);
    float middle = energies.get(i, min_index);
    float right = energies.get(i, min_index + 1);

    // Have the seam follow the least cost.
    if (left < middle && left < right) {
      min_index--; // go left
    } else if (right < middle && right < left) {
      min_index++; // go right
    }

    // Append to the seam.
    _seam.push_back(min_index);
  }

  // Clean up.
  std::reverse(_seam.begin(), _seam.end());
}


float Seamcarver::minCost(Energies& energies, int i, int j) {
  // For top row entries we return the element itself.
  if (i <= 0) {
    return energies.get(i, j);
  }

  // Take the 3 adjacent cells in the above row.
  float left = energies.get(i - 1, j - 1);
  float middle = energies.get(i - 1, j);
  float right = energies.get(i - 1, j + 1);

  // Compute the minimum, add cost of current cell.
  float minimum = min(left, min(middle, right));
  return minimum + energies.get(i, j);
}


int Seamcarver::nextPower2(int n) {
  n--;
  n = n >>  1 | n;
  n = n >>  2 | n;
  n = n >>  4 | n;
  n = n >>  8 | n;
  n = n >> 16 | n;
  return ++n;
}
